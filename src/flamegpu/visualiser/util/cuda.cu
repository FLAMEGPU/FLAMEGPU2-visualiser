#include "flamegpu/visualiser/util/hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include <>

#include <cstring>
#include <typeinfo>   //  operator typeid
#include <stdexcept>


namespace flamegpu {
namespace visualiser {

// Drop in replacement if CUDA_CALL is missing
#ifndef CUDA_CALL
#define CUDA_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        // TODO: Exception with message?
        // THROW CUDAError("CUDA Error: %s(%d): %s", file, line, hipGetErrorString(code));
        fprintf(stdout, "CUDA Error: %s(%d): %s", file, line, hipGetErrorString(code));
        throw std::exception();
    }
}
#endif


/*
Hidden internal functions
*/
namespace {
/*
Internal function used to detect the required internal format
@param componentCount The number of components
@param a A garbage value to specify the desired type because specialised templates werent working
@return the internal format
@see https:// www.opengl.org/sdk/docs/man/html/glTexBuffer.xhtml
*/
GLenum _getInternalFormat(const unsigned int componentCount, float) {
    if (componentCount == 1) return GL_R32F;
    if (componentCount == 2) return GL_RG32F;
    if (componentCount == 3 || componentCount == 4) return GL_RGBA32F;
    return 0;
}
GLenum _getInternalFormat(const unsigned int componentCount, unsigned int) {
    if (componentCount == 1) return GL_R32UI;
    if (componentCount == 2) return GL_RG32UI;
    if (componentCount == 3 || componentCount == 4) return GL_RGBA32UI;
    return 0;
}
GLenum _getInternalFormat(const unsigned int componentCount, int) {
    if (componentCount == 1) return GL_R32I;
    if (componentCount == 2) return GL_RG32I;
    if (componentCount == 3 || componentCount == 4) return GL_RGBA32I;
    return 0;
}
/*
@param componentCount The number of components (1-2, 4). Passing 3 will be treated as 4
@param bufferSize The total size of the buffer in bytes
@param d_TexPointer A device pointer to the mapped texture buffer
@return The filled cudaResourDesc (memset to 0 if invalid inputs)
*/
template<class T>
hipResourceDesc _getCUDAResourceDesc(const unsigned int componentCount, const unsigned int bufferSize, const T *d_TexPointer) {
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    // Return empty if invalid input
    if (d_TexPointer == 0 ||
        bufferSize == 0 ||
        componentCount == 0 ||
        componentCount > 4)
        return resDesc;
    // Linear because its a texture buffer, not a texture
    resDesc.resType = hipResourceTypeLinear;
    // Mapped pointer to the texture buffer on the device
    resDesc.res.linear.devPtr = reinterpret_cast<void*>(const_cast<T*>(d_TexPointer));
    // The type of the components

    if (typeid(T) == typeid(float)) {
        resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    } else if (typeid(T) == typeid(unsigned int)) {
        resDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
    } else if (typeid(T) == typeid(int)) {
        resDesc.res.linear.desc.f = hipChannelFormatKindSigned;
    }
    // The number of bits per component (0 if not used)
    resDesc.res.linear.desc.x = 32;
    if (componentCount >= 2) {
        resDesc.res.linear.desc.y = 32;
        if (componentCount >= 3) {
            resDesc.res.linear.desc.z = 32;
            resDesc.res.linear.desc.w = 32;
        }
    }
    // The total buffer size
    resDesc.res.linear.sizeInBytes = bufferSize;
    return resDesc;
}
}  // namespace

template<class T>
CUDATextureBuffer<T> *mallocGLInteropTextureBuffer(const unsigned int elementCount, const unsigned int t_componentCount) {
    if (elementCount == 0||
        t_componentCount == 0 ||
        t_componentCount > 4)
        return nullptr;
    // Temporary storage of return values
    GLuint glTexName;
    GLuint glTBO;
    T *d_MappedPointer = nullptr;
    hipGraphicsResource_t cuGraphicsRes;
    hipTextureObject_t cuTextureObj;

    // Interpretation of buffer type/component details
    const unsigned int componentCount = t_componentCount == 3 ? 4 : t_componentCount;
    const unsigned int componentSize = sizeof(T);
    const unsigned int elementSize = componentSize*componentCount;
    const unsigned int bufferSize = elementSize * elementCount;
    const GLuint internalFormat = _getInternalFormat(componentCount, static_cast<T>(0));

    // Gen tex
    GL_CALL(glGenTextures(1, &glTexName));
    // Gen buffer
    GL_CALL(glGenBuffers(1, &glTBO));
    // Size buffer and tie to tex
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, glTBO));
    GL_CALL(glBufferData(GL_TEXTURE_BUFFER, bufferSize, 0, GL_STATIC_DRAW));                                    // TODO dynamic draw better?

    GL_CALL(glBindTexture(GL_TEXTURE_BUFFER, glTexName));
    GL_CALL(glTexBuffer(GL_TEXTURE_BUFFER, internalFormat, glTBO));
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, 0));
    GL_CALL(glBindTexture(GL_TEXTURE_BUFFER, 0));

    // Get CUDA handle to texture
    CUDA_CALL(hipGraphicsGLRegisterBuffer(&cuGraphicsRes, glTBO, cudaGraphicsMapFlagsNone));
    // Map/convert this to something cuGraphicsRes
    CUDA_CALL(hipGraphicsMapResources(1, &cuGraphicsRes));
    CUDA_CALL(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_MappedPointer), 0, cuGraphicsRes));
    CUDA_CALL(hipGraphicsUnmapResources(1, &cuGraphicsRes, 0));
    // Create a texture object from the cuGraphicsRes
    hipResourceDesc resDesc = _getCUDAResourceDesc(componentCount, bufferSize, d_MappedPointer);
    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.readMode = hipReadModeElementType;  // Read as actual type, other option is normalised float
    // texDesc.addressMode[0] = hipAddressModeWrap;  // We can only affect the address mode for first 3 dimensions, so lets leave it default
    CUDA_CALL(hipCreateTextureObject(&cuTextureObj, &resDesc, &texDesc, nullptr));
    // Copy the generated data
    return new CUDATextureBuffer<T>(glTexName, glTBO, d_MappedPointer, cuGraphicsRes, cuTextureObj, elementCount, componentCount);
}
template<class T>
void freeGLInteropTextureBuffer(CUDATextureBuffer<T> *texBuf) {
    CUDA_CALL(hipDestroyTextureObject(texBuf->cuTextureObj));
    CUDA_CALL(hipGraphicsUnregisterResource(texBuf->cuGraphicsRes));
    GL_CALL(glDeleteBuffers(1, &texBuf->glTBO));
    GL_CALL(glDeleteTextures(1, &texBuf->glTexName));
    delete texBuf;
}
/**
 * Returns true if hipMemcpy returns hipSuccess
 */
bool _cudaMemcpyDeviceToDevice(void* dst, const void* src, size_t count) {
    auto t = hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice);
    CUDA_CALL(t);
    return t == hipSuccess;
}
// Explicit instantiation of templates
template CUDATextureBuffer<float> *mallocGLInteropTextureBuffer(const unsigned int, const unsigned int);
template CUDATextureBuffer<int> *mallocGLInteropTextureBuffer(const unsigned int, const unsigned int);
template CUDATextureBuffer<unsigned int> *mallocGLInteropTextureBuffer(const unsigned int, const unsigned int);
template void freeGLInteropTextureBuffer(CUDATextureBuffer<float>*);
template void freeGLInteropTextureBuffer(CUDATextureBuffer<int>*);
template void freeGLInteropTextureBuffer(CUDATextureBuffer<unsigned int>*);

}  // namespace visualiser
}  // namespace flamegpu
